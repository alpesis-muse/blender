#include "hip/hip_runtime.h"
#define tx threadIdx.x
#define ty threadIdx.y
#define bx blockIdx.x
#define by blockIdx.y
#define bdx blockDim.x
#define bdy blockDim.y

#define MAX_KSIZE_HALF 100

__constant__ float c_gKer[MAX_KSIZE_HALF+1];

__global__ void gaussian_blur_kernel (const int height,
                                      const int width,
                                      const int center,
                                      const T * src,
                                      const T * dst)
{
	const int y = by * bdy + ty;
	const int x = bx * bdx + tx;
	extern __shared__ float smem[];
	volatile float * row = smem + ty * (bdx + 2 * center);
	
	if (y < height)
	{
		for (int i = tx; i < bdx + 2 * center; i += bdx)
		{
			int x_ext = int(bx * bdx) + i - center;
			x_ext = b.idx_col(x_ext);
			row[i] = src(y, x_ext) * c_gKer[0];
			for (int j = 1; j <= center; ++j)
			{
				row[i] += (src(b.idx_row_low(y-j), x_ext) +
                                           src(b.idx_row_high(y+j), x_ext)) * c_gKer[j];
			}
		}

		if (x < width)
		{
			__syncthreads();
			// horizontal pass
			row += tx + center;
			float res = row[0] * c_gKer[0];
			for (int i = 1; i <= center; ++i)
			{
				res += (row[-i] + row[i]) * c_gKer[i];
			}
			dst(y, x) = res;
		}
	} 
}
